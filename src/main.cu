#include "hip/hip_runtime.h"
#include "utility.h"

#include "color.h"
#include "hittable_list.h"
#include "sphere.h"
#include "mesh.h"

#include <iostream>

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ float ray_intensity(const ray r, const hittable_list world) {
    hit_record rec;
      if (world.hit(r, 0, infinity, rec)) {
            return rec.trans_prob; // if hit, return the probability of transmission
        }
      else {
          return 1.0f; // if not hit, return 1 (vacuum)
      }
}
__global__ void render(float *fb, int image_width, int image_height, hittable_list **world, vec3 origin, vec3 lower_left_corner, vec3 horizontal, vec3 vertical) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= image_width) || (j >= image_height)) return;
    int pixel_index = j * image_width + i;
    float u = float(i) / (image_width - 1);
    float v = float(j) / (image_height - 1);
    ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = ray_intensity(r, **world);
}

__global__ void create_world(hittable_list **world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        material bone_mtr = material(3.148E-01, 1.8); // material of the bone
        mesh mesh1 = mesh("stl/ancient_chinese_coin.stl", vec3(0, 0, 0), device_ptr<material>(&bone_mtr)); // create the mesh
        (**world).add(device_ptr<hittable>(&mesh1)); // add the mesh to the world
    }
}



int main() {

    // Image
    const float aspect_ratio = 16.0 / 9.0;
    const int image_width = 800;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    const int thread_width = 8;
    const int thread_height = 8;
    const int num_pixels = image_width * image_height;

    size_t fb_size = num_pixels * sizeof(float); // framebuffer size
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size)); // allocate framebuffer

    // World
    hittable_list **d_world; // list of objects in the world;
    checkCudaErrors(hipMallocManaged((void **)&d_world, sizeof(hittable))); // allocate memory for the world
    create_world<<<1, 1>>>(d_world); // create the world
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // Camera
    const float viewport_height = 12.0;
    const float viewport_width = aspect_ratio * viewport_height;
    const float focal_length = 6.0;

    vec3 origin = vec3(0, 0, 0);
    vec3 horizontal = vec3(viewport_width, 0, 0);
    vec3 vertical = vec3(0, viewport_height, 0);
    vec3 lower_left_corner = origin - horizontal/2 - vertical/2 - vec3(-1, 0, focal_length);

    // Render our buffer
    dim3 blocks(image_width/thread_width + 1, image_height/thread_height + 1);
    dim3 threads(thread_width, thread_height);
    render<<<blocks, threads>>>(fb, image_width, image_height, d_world, origin, lower_left_corner, horizontal, vertical);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output FB as Image
    std::ofstream render;
    render.open("examples/ancient_chinese_coin.pgm"); // open pgm file for writing greyscale image
    render << "P2\n" << image_width << ' ' << image_height << "\n255\n";
    for (int j = image_height-1; j >= 0; --j) {
        std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; ++i) {
            size_t pixel_index = j*image_width+ i;
            write_color(render, fb[pixel_index]);
        }
    }
    render.close();
    std::cerr << "\nDone.\n";

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(&d_world));



    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
    return 0;
}